#include "hip/hip_runtime.h"
#include <stdio.h>

#include "kernel.hpp"
#include "model.hpp"


/******************************************************************************
Constants
******************************************************************************/


/* leaky_relu activation slope */
const float LEAKY_RELU_SLOPE = .2;

/* amount of reflection padding per side */
const unsigned int REFLECTION_PAD_SIZE = 3;


/******************************************************************************
Kernels
******************************************************************************/


namespace kernel {
    /* addition */
    __global__ void add(float *x,
                        const float * const y,
                        const unsigned int size)
    {
        unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index < size) x[index] += y[index];
    }

    __global__ void broadcast_add(float *x,
                                  const float * const y,
                                  const unsigned int rows,
                                  const unsigned int cols)
    {

    }


    /* leaky relu activation */
    __global__ void leaky_relu(float *input, const unsigned int size)
    {
        const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index < size) {
            const float value = input[index];
            input[index] = fmaxf(value, LEAKY_RELU_SLOPE * value);
        }
    }


    /* printing utility */
    __global__ void print(float *input, const unsigned int size)
    {
        if (blockIdx.x * blockDim.x + threadIdx.x == 0) {
            for (unsigned int i = 1000000; i < 1000000 + size; ++i) printf("%f ", input[i]);
            printf("\n");
        }
    }


    /* reflection padding */
    __global__ void reflection_padding(float *input,
                                       float *output,
                                       const unsigned int frames,
                                       const unsigned int channels,
                                       const unsigned int padding)
    {
        const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
        const unsigned int output_frames = frames + 2 * padding;
        const unsigned int channel = index / output_frames;
        const int output_frame = index % output_frames;
        int input_frame = (output_frame - padding);

        if (index < output_frames * channels) {
            // Reflect
            if (input_frame < 0)
                input_frame = -input_frame;
            else if (input_frame >= frames)
                input_frame = frames - (input_frame - frames) - 2;

            // Pad
            output[channel * output_frames + output_frame] =
                input[channel * frames + input_frame];
        }
    }


    /* tanh activation */
    __global__ void tanh(float *input, const unsigned int size)
    {
        const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index < size) input[index] = tanhf(input[index]);
    }
}
