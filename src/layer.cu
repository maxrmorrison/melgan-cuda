#include "hip/hip_runtime.h"
#include <hipDNN.h>
#include <iostream>
#include <stdio.h>

#include "cuda.hpp"
#include "layer.hpp"
#include "kernel.hpp"


/******************************************************************************
Utilities
******************************************************************************/


static int checkCudnnError(hipdnnStatus_t code,
                           const char *expr,
                           const char *file,
                           int line)
{
    if (code)
    {
        printf(
            "CUDNN error at %s:%d, code=%d (%s) in '%s'\n",
            file,
            line,
            (int)code,
            hipdnnGetErrorString(code),
            expr);
        return 1;
    }
    return 0;
}


#define checkCudnnErr(...)                                  \
    do                                                      \
    {                                                       \
        int err = checkCudnnError(                          \
            __VA_ARGS__, #__VA_ARGS__, __FILE__, __LINE__); \
        if (err) exit(1);                                   \
    } while (0)


unsigned int get_num_output_frames_backward(unsigned int input_frames,
                                            const Convolution &convolution)
{
    unsigned int d = convolution.dilation;
    unsigned int k = convolution.kernel_size;
    unsigned int p = convolution.zero_padding;
    unsigned int s = convolution.stride;
    return (input_frames - 1) * s - 2 * p + d * (k - 1) + 1;
}


unsigned int get_num_output_frames_forward(unsigned int input_frames,
                                           const Convolution &convolution)
{
    unsigned int d = convolution.dilation;
    unsigned int k = convolution.kernel_size;
    unsigned int p = convolution.zero_padding;
    unsigned int s = convolution.stride;
    return ((input_frames + 2 * p - d * (k - 1) - 1) / (float) s + 1);
}


/******************************************************************************
Constants
******************************************************************************/


const unsigned int THREADS_PER_BLOCK = 1024;


/******************************************************************************
Layers
******************************************************************************/


namespace layer {
    /* addition */
    float *add(float *x, float *y, const unsigned int size)
    {
        // Add in-place
        const unsigned int blocks = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        kernel::add<<<blocks, THREADS_PER_BLOCK>>>(x, y, size);
        hipDeviceSynchronize();

        // Free y
        hipFree(y);

        // User frees x
        return x;
    }


    /* convolution */
    float *conv(float *input,
                const unsigned int frames,
                const Convolution &convolution,
                hipdnnHandle_t cudnn,
                bool free_input)
    {
        unsigned int output_frames = get_num_output_frames_forward(
            frames, convolution);

        // Setup input
        hipdnnTensorDescriptor_t input_descriptor;
        checkCudnnErr(hipdnnCreateTensorDescriptor(&input_descriptor));
        checkCudnnErr(hipdnnSetTensor4dDescriptor(
            input_descriptor,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/convolution.input_channels,
            /*image_height=*/1,
            /*image_width=*/frames));

        // Setup output
        hipdnnTensorDescriptor_t output_descriptor;
        checkCudnnErr(hipdnnCreateTensorDescriptor(&output_descriptor));
        checkCudnnErr(hipdnnSetTensor4dDescriptor(
            output_descriptor,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/convolution.output_channels,
            /*image_height=*/1,
            /*image_width=*/output_frames));

        // Setup kernel
        hipdnnFilterDescriptor_t kernel_descriptor;
        checkCudnnErr(hipdnnCreateFilterDescriptor(&kernel_descriptor));
        checkCudnnErr(hipdnnSetFilter4dDescriptor(
            kernel_descriptor,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*out_channels=*/convolution.output_channels,
            /*in_channels=*/convolution.input_channels,
            /*kernel_height=*/1,
            /*kernel_width=*/convolution.kernel_size));

        // Setup bias
        hipdnnTensorDescriptor_t bias_descriptor;
        checkCudnnErr(hipdnnCreateTensorDescriptor(&bias_descriptor));
        checkCudnnErr(hipdnnSetTensor4dDescriptor(
            bias_descriptor,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/convolution.output_channels,
            /*image_height=*/1,
            /*image_width=*/1));

        // Setup convolution
        hipdnnConvolutionDescriptor_t convolution_descriptor;
        checkCudnnErr(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
        checkCudnnErr(hipdnnSetConvolution2dDescriptor(
            convolution_descriptor,
            /*pad_height=*/0,
            /*pad_width=*/convolution.zero_padding,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/convolution.stride,
            /*dilation_height=*/1,
            /*dilation_width=*/convolution.dilation,
            /*mode=*/HIPDNN_CROSS_CORRELATION,
            /*computeType=*/HIPDNN_DATA_FLOAT));
        checkCudnnErr(hipdnnSetConvolutionMathType(
            convolution_descriptor,
            CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION));

        // Setup workspace
        hipdnnConvolutionFwdAlgo_t convolution_algorithm =
            HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
        size_t workspace_bytes = 0;
        checkCudnnErr(hipdnnGetConvolutionForwardWorkspaceSize(
            cudnn,
            input_descriptor,
            kernel_descriptor,
            convolution_descriptor,
            output_descriptor,
            convolution_algorithm,
            &workspace_bytes));
        void *workspace = nullptr;
        hipMalloc(&workspace, workspace_bytes);

        // Setup activation
        hipdnnActivationDescriptor_t activation_descriptor;
        checkCudnnErr(hipdnnCreateActivationDescriptor(&activation_descriptor));
        checkCudnnErr(hipdnnSetActivationDescriptor(
            activation_descriptor,
            HIPDNN_ACTIVATION_PATHTRU,
            HIPDNN_PROPAGATE_NAN,
            0.));

        // Allocate output
        float *output = cuda::allocate(
            convolution.output_channels * output_frames * sizeof(float));

        // No blending
        float alpha1 = 1.0f, alpha2 = 0.0f;

        // Perform op
        checkCudnnErr(cudnnConvolutionBiasActivationForward(
            cudnn,
            &alpha1,
            input_descriptor,
            input,
            kernel_descriptor,
            convolution.weight_d,
            convolution_descriptor,
            convolution_algorithm,
            workspace,
            workspace_bytes,
            &alpha2,
            output_descriptor,
            output,
            bias_descriptor,
            convolution.bias_d,
            activation_descriptor,
            output_descriptor,
            output));

        // Clean up
        hipdnnDestroyActivationDescriptor(activation_descriptor);
        hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
        hipdnnDestroyFilterDescriptor(kernel_descriptor);
        hipdnnDestroyTensorDescriptor(input_descriptor);
        hipdnnDestroyTensorDescriptor(output_descriptor);
        hipdnnDestroyTensorDescriptor(bias_descriptor);
        hipFree(workspace);

        // Optionally free input
        if (free_input) cuda::free(input);

        // User frees output
        return output;
    }


    /* leaky relu activation */
    float *leaky_relu(float *activation, const unsigned int size)
    {
        const unsigned int blocks =
            (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        kernel::leaky_relu<<<blocks, THREADS_PER_BLOCK>>>(activation, size);
        hipDeviceSynchronize();
        return activation;
    }


    /* printing utility */
    void print(float *activation, const unsigned int size)
    {
        kernel::print<<<1, 1>>>(activation, size);
        hipDeviceSynchronize();
    }


    float *reflection_padding(float *activation,
                              const unsigned int frames,
                              const unsigned int channels,
                              const unsigned int padding,
                              bool free_input)
    {
        // Allocate output
        const unsigned int output_frames = frames + 2 * padding;
        const unsigned int output_size = output_frames * channels;
        const unsigned int output_bytes = output_size * sizeof(float);
        float *output = cuda::allocate(output_bytes);

        // Perform padding
        const unsigned int blocks =
            (output_size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        kernel::reflection_padding<<<blocks, THREADS_PER_BLOCK>>>(
            activation, output, frames, channels, padding);
        hipDeviceSynchronize();

        // Optionally free input
        if (free_input) cuda::free(activation);

        // User frees output
        return output;
    }


    /* tanh activation */
    float *tanh(float *activation, const unsigned int size)
    {
        const unsigned int blocks =
            (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        kernel::tanh<<<blocks, THREADS_PER_BLOCK>>>(activation, size);
        hipDeviceSynchronize();
        return activation;
    }


    /* transpose convolution */
    float *transpose_conv(float *input,
                          const unsigned int frames,
                          const Convolution &convolution,
                          hipdnnHandle_t cudnn,
                          bool free_input)
    {
        unsigned int output_frames = get_num_output_frames_backward(
            frames, convolution);

        // Setup input
        hipdnnTensorDescriptor_t input_descriptor;
        checkCudnnErr(hipdnnCreateTensorDescriptor(&input_descriptor));
        checkCudnnErr(hipdnnSetTensor4dDescriptor(
            input_descriptor,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/convolution.output_channels,
            /*image_height=*/1,
            /*image_width=*/frames));

        // Setup output
        hipdnnTensorDescriptor_t output_descriptor;
        checkCudnnErr(hipdnnCreateTensorDescriptor(&output_descriptor));
        checkCudnnErr(hipdnnSetTensor4dDescriptor(
            output_descriptor,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/convolution.input_channels,
            /*image_height=*/1,
            /*image_width=*/output_frames));

        // Setup kernel
        hipdnnFilterDescriptor_t kernel_descriptor;
        checkCudnnErr(hipdnnCreateFilterDescriptor(&kernel_descriptor));
        checkCudnnErr(hipdnnSetFilter4dDescriptor(
            kernel_descriptor,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*out_channels=*/convolution.output_channels,
            /*in_channels=*/convolution.input_channels,
            /*kernel_height=*/1,
            /*kernel_width=*/convolution.kernel_size));

        // Setup bias
        hipdnnTensorDescriptor_t bias_descriptor;
        checkCudnnErr(hipdnnCreateTensorDescriptor(&bias_descriptor));
        checkCudnnErr(hipdnnSetTensor4dDescriptor(
            bias_descriptor,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/convolution.input_channels,
            /*image_height=*/1,
            /*image_width=*/1));

        // Setup convolution
        hipdnnConvolutionDescriptor_t convolution_descriptor;
        checkCudnnErr(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
        checkCudnnErr(hipdnnSetConvolution2dDescriptor(
            convolution_descriptor,
            /*pad_height=*/0,
            /*pad_width=*/convolution.zero_padding,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/convolution.stride,
            /*dilation_height=*/1,
            /*dilation_width=*/convolution.dilation,
            /*mode=*/HIPDNN_CROSS_CORRELATION,
            /*computeType=*/HIPDNN_DATA_FLOAT));

        // Setup workspace
        hipdnnConvolutionBwdDataAlgo_t convolution_algorithm =
            HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0;
        size_t workspace_bytes = 0;
        checkCudnnErr(hipdnnGetConvolutionBackwardDataWorkspaceSize(
            cudnn,
            kernel_descriptor,
            input_descriptor,
            convolution_descriptor,
            output_descriptor,
            convolution_algorithm,
            &workspace_bytes));
        void *workspace = nullptr;
        hipMalloc(&workspace, workspace_bytes);

        // Allocate output
        float *output = cuda::allocate(
            convolution.input_channels * output_frames * sizeof(float));

        // No blending
        float alpha = 1.0f, beta = 0.0f;

        // Perform op
        checkCudnnErr(hipdnnConvolutionBackwardData(
            cudnn,
            &alpha,
            kernel_descriptor,
            convolution.weight_d,
            input_descriptor,
            input,
            convolution_descriptor,
            convolution_algorithm,
            workspace,
            workspace_bytes,
            &beta,
            output_descriptor,
            output));
        beta = 1.;
        checkCudnnErr(hipdnnAddTensor(
            cudnn,
            &alpha,
            bias_descriptor,
            convolution.bias_d,
            &beta,
            output_descriptor,
            output));

        // Clean up
        hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
        hipdnnDestroyFilterDescriptor(kernel_descriptor);
        hipdnnDestroyTensorDescriptor(input_descriptor);
        hipdnnDestroyTensorDescriptor(output_descriptor);
        hipdnnDestroyTensorDescriptor(bias_descriptor);
        hipFree(workspace);

        // Optionally free input
        if (free_input) cuda::free(input);

        // User frees output
        return output;
        }
    }
