#include "cuda.hpp"


namespace cuda {
    /* allocate memory */
    float *allocate(const unsigned int size)
    {
        float *data;
        hipMalloc((void **) &data, size);
        return data;
    }


    /* copy memory to device */
    void copy_to_device(float *device,
                        const float *host,
                        const unsigned int size)
    {
        hipMemcpy(device, host, size, hipMemcpyHostToDevice);
    }


    /* copy memory to host */
    void copy_to_host(float *host,
                      const float *device,
                      const unsigned int size)
    {
        hipMemcpy(host, device, size, hipMemcpyDeviceToHost);
    }


    /* free memory */
    void free(float *data)
    {
        hipFree(data);
    }
}
